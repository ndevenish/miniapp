#include "hip/hip_runtime.h"
/**
 * Basic Naive Kernel
 * 
 * Does spotfinding in-kernel, without in-depth performance tweaking.
 * 
 */

#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include <cooperative_groups/scan.h>
#include <fmt/core.h>

#include <array>
#include <cassert>
#include <chrono>
#include <memory>
#include <utility>

#include "common.hpp"
#include "h5read.h"
#include "standalone.h"

namespace cg = cooperative_groups;

using namespace fmt;

using pixel_t = H5Read::image_type;

/// One-direction width of kernel. Total kernel span is (K_W * 2 + 1)
constexpr int KERNEL_WIDTH = 3;
/// One-direction height of kernel. Total kernel span is (K_H * 2 + 1)
constexpr int KERNEL_HEIGHT = 3;

__global__ void do_spotfinding_naive(PitchedMemoryArea<pixel_t> image,
                                     PitchedMemoryArea<uint8_t> mask,
                                     int width,
                                     int height,
                                     PitchedMemoryArea<int> result_sum,
                                     PitchedMemoryArea<size_t> result_sumsq,
                                     PitchedMemoryArea<uint8_t> result_n,
                                     PitchedMemoryArea<uint8_t> result_strong) {
    const size_t image_pitch = image.pitch;
    const size_t mask_pitch = mask.pitch;

    __shared__ uint64_t block_exchange_8[32][32];
    __shared__ uint32_t block_exchange_4[32][32];
    __shared__ uint8_t block_exchange_1[32][32];

    auto block = cg::this_thread_block();
    auto warp = cg::tiled_partition<32>(block);
    // int warpId = warp.meta_group_rank();
    // int lane = warp.thread_rank();

    uint sum = 0;
    size_t sumsq = 0;
    uint8_t n = 0;

    // The target image pixel of this thread
    int x = block.group_index().x * block.group_dim().x + block.thread_index().x
            - KERNEL_WIDTH;
    int y = block.group_index().y * block.group_dim().y + block.thread_index().y
            - KERNEL_HEIGHT;

    // Make sure this pixel isn't masked or off-image
    bool px_is_valid = false;
    pixel_t this_pixel = 0;
    if (x >= 0 && y >= 0 && x < width && y < height) [[likely]] {
        px_is_valid = mask[y * mask_pitch + x] != 0;
        this_pixel = image[y * image_pitch + x];
    }
    size_t this_pixel_sq = this_pixel * this_pixel;
    if (y == 1 && x == 2) {
        printf("%d, %d: pixel = %d, mask = %s\n",
               x,
               y,
               (int)this_pixel,
               px_is_valid ? "true" : "false");
    }

    // Calculate the horizontal prefix sums
    auto inc_sum = cg::inclusive_scan(warp, this_pixel);
    auto inc_sumsq = cg::inclusive_scan(warp, this_pixel_sq);
    auto inc_n = cg::inclusive_scan(warp, px_is_valid ? 1 : 0);

    if (y == 1 && x == 2) {
        printf("%d, %d: sum = %d, sq = %d, n = %d\n",
               x,
               y,
               (int)inc_sum,
               (int)inc_sumsq,
               (int)inc_n);
    }

    // Broadcast this value to the rest of the block
    block_exchange_8[block.thread_index().y][block.thread_index().x] = inc_sumsq;
    block_exchange_4[block.thread_index().y][block.thread_index().x] = inc_sum;
    block_exchange_1[block.thread_index().y][block.thread_index().x] = inc_n;

    // Wait until we can do block-level exchanges
    __syncthreads();

    // Transpose the block
    inc_sumsq = block_exchange_8[block.thread_index().x][block.thread_index().y];
    inc_sum = block_exchange_4[block.thread_index().x][block.thread_index().y];
    inc_n = block_exchange_1[block.thread_index().x][block.thread_index().y];

    if (y == 1 && x == 2) {
        printf("%d, %d: sum = %d, sq = %d, n = %d After reading transpose\n",
               x,
               y,
               (int)inc_sum,
               (int)inc_sumsq,
               (int)inc_n);
    }

    __syncthreads();
    inc_sumsq = cg::inclusive_scan(warp, inc_sumsq);
    inc_sum = cg::inclusive_scan(warp, inc_sum);
    inc_n = cg::inclusive_scan(warp, inc_n);

    if (y == 1 && x == 2) {
        printf("%d, %d: sum = %d, sq = %d, n = %d After summing transpose\n",
               x,
               y,
               (int)inc_sum,
               (int)inc_sumsq,
               (int)inc_n);
    }
    // And, write it back
    block_exchange_8[block.thread_index().y][block.thread_index().x] = inc_sumsq;
    block_exchange_4[block.thread_index().y][block.thread_index().x] = inc_sum;
    block_exchange_1[block.thread_index().y][block.thread_index().x] = inc_n;

    __syncthreads();
    // Now, pull down the incremental sum for this pixel again
    inc_sumsq = block_exchange_8[block.thread_index().y][block.thread_index().x];
    inc_sum = block_exchange_4[block.thread_index().y][block.thread_index().x];
    inc_n = block_exchange_1[block.thread_index().y][block.thread_index().x];

    if (y == 1 && x == 2) {
        printf("%d, %d: sum = %d, sq = %d, n = %d after reP-reading\n",
               x,
               y,
               (int)inc_sum,
               (int)inc_sumsq,
               (int)inc_n);
    }
    if (x >= 0 && y >= 0 && x < width && y < height) {
        result_sum[y * image_pitch + x] = inc_sum;
        result_sumsq[y * image_pitch + x] = inc_sumsq;
        result_n[y * mask_pitch + x] = inc_n;
    } else if (x < 0 || y < 0) {
        // Don't try to access memory if out-of-bounds
        return;
    }

    if (px_is_valid) {
        for (int row = max(0, y - KERNEL_HEIGHT);
             row < min(y + KERNEL_HEIGHT + 1, height);
             ++row) {
            int row_offset = image_pitch * row;
            int mask_offset = mask_pitch * row;
            for (int col = max(0, x - KERNEL_WIDTH);
                 col < min(x + KERNEL_WIDTH + 1, width);
                 ++col) {
                pixel_t pixel = image[row_offset + col];
                uint8_t mask_pixel = mask[mask_offset + col];
                if (mask_pixel) {
                    sum += pixel;
                    sumsq += pixel * pixel;
                    n += 1;
                }
            }
        }
    }

    if (x < width && y < height && x >= 0 && y >= 0) {
        // result_sum[x + image_pitch * y] = sum;
        // result_sumsq[x + image_pitch * y] = sumsq;
        // result_n[x + mask_pitch * y] = n;

        // Calculate the thresholding
        if (px_is_valid) {
            constexpr float n_sig_s = 3.0f;
            constexpr float n_sig_b = 6.0f;

            float sum_f = static_cast<float>(sum);
            float sumsq_f = static_cast<float>(sumsq);

            float mean = sum_f / n;
            float variance = (n * sumsq_f - (sum_f * sum_f)) / (n * (n - 1));
            float dispersion = variance / mean;
            float background_threshold = 1 + n_sig_b * sqrt(2.0f / (n - 1));
            bool not_background = dispersion > background_threshold;
            float signal_threshold = mean + n_sig_s * sqrt(mean);
            bool is_signal = this_pixel > signal_threshold;
            bool is_strong_pixel = not_background && is_signal;
            result_strong[x + mask_pitch * y] = is_strong_pixel;
        } else {
            result_strong[x + mask_pitch * y] = 0;
        }
    }
}

int main(int argc, char **argv) {
    // Parse arguments and get our H5Reader
    auto parser = CUDAArgumentParser();
    parser.add_h5read_arguments();
    auto args = parser.parse_args(argc, argv);

    auto reader = args.file.empty() ? H5Read() : H5Read(args.file);

    int height = reader.image_shape()[0];
    int width = reader.image_shape()[1];

    // Work out how many blocks this is
    dim3 thread_block_size{32, 32};
    // Make enough blocks to overlap the edges with the kernel
    dim3 blocks_dims{
      static_cast<unsigned int>(
        ceilf(static_cast<float>(width + KERNEL_WIDTH * 2) / thread_block_size.x)),
      static_cast<unsigned int>(
        ceilf(static_cast<float>(height + KERNEL_HEIGHT * 2) / thread_block_size.y))};
    const int num_threads_per_block = thread_block_size.x * thread_block_size.y;
    const int num_blocks = blocks_dims.x * blocks_dims.y;
    print("Image:   {:4d} x {:4d} = {} px\n", width, height, width * height);
    print("Threads: {:4d} x {:<4d} = {}\n",
          thread_block_size.x,
          thread_block_size.y,
          num_threads_per_block);
    print("Blocks:  {:4d} x {:<4d} = {}\n", blocks_dims.x, blocks_dims.y, num_blocks);

    // Create a host memory area to read the image into
    // auto host_image = std::make_unique<pixel_t[]>(width * height);
    auto host_image = make_cuda_pinned_malloc<pixel_t>(width * height);

    // Device-side pitched storage for image data
    auto dev_image = make_cuda_pitched_malloc<pixel_t>(width, height);
    auto dev_mask = make_cuda_pitched_malloc<uint8_t>(width, height);
    print("Allocated device memory. Pitch = {} vs naive {}\n", dev_image.pitch, width);

    // Managed memory areas for results
    auto result_sum =
      HostPitchedMemoryArea(dev_image.pitch,
                            width,
                            height,
                            make_cuda_managed_malloc<int>(dev_image.pitch * height));
    auto result_sumsq =
      HostPitchedMemoryArea(dev_image.pitch,
                            width,
                            height,
                            make_cuda_managed_malloc<size_t>(dev_image.pitch * height));
    auto result_n =
      HostPitchedMemoryArea(dev_mask.pitch,
                            width,
                            height,
                            make_cuda_managed_malloc<uint8_t>(dev_mask.pitch * height));
    auto result_strong =
      HostPitchedMemoryArea(dev_mask.pitch,
                            width,
                            height,
                            make_cuda_managed_malloc<uint8_t>(dev_mask.pitch * height));
    // Make sure to clear these completely
    hipMemset(result_sum.get(), 0, dev_image.bytes());
    hipMemset(result_sumsq.get(), 0, sizeof(size_t) * dev_image.pitch * height);
    hipMemset(result_n.get(), 0, sizeof(uint8_t) * dev_mask.pitch * height);
    hipMemset(result_strong.get(), 0, sizeof(uint8_t) * dev_mask.pitch * height);
    hipDeviceSynchronize();
    cuda_throw_error();

    CudaEvent pre_load, start, memcpy, kernel, all;

    size_t mask_sum = 0;
    if (reader.get_mask()) {
        mask_sum = 0;
        for (size_t i = 0; i < width * height; ++i) {
            if (reader.get_mask().value()[i]) {
                mask_sum += 1;
            }
        }
        start.record();
        hipMemcpy2D(dev_mask.get(),
                     dev_mask.pitch,
                     reader.get_mask()->data(),
                     width,
                     width,
                     height,
                     hipMemcpyHostToDevice);
        cuda_throw_error();
    } else {
        mask_sum = width * height;
        start.record();
        hipMemset(dev_mask.get(), 1, dev_mask.pitch * height);
        cuda_throw_error();
    }
    memcpy.record();
    memcpy.synchronize();

    float memcpy_time = memcpy.elapsed_time(start);
    print("Uploaded mask ({:.2f} Mpx) in {:.2f} ms ({:.1f} GBps)\n",
          static_cast<float>(mask_sum) / 1e6,
          memcpy_time,
          GBps(memcpy_time, width * height));

    print("\nProcessing {} Images\n\n", reader.get_number_of_images());
    auto spotfinder = StandaloneSpotfinder(width, height);

    for (size_t image_id = 0; image_id < reader.get_number_of_images(); ++image_id) {
        if (args.image_number.has_value() && args.image_number.value() != image_id) {
            continue;
        }

        print("Image {}:\n", image_id);
        pre_load.record();
        pre_load.synchronize();

        reader.get_image_into(image_id, host_image.get());

        // Copy data to GPU
        // Copy the image to GPU
        start.record();
        hipMemcpy2D(dev_image.get(),
                     dev_image.pitch_bytes(),
                     host_image.get(),
                     width * sizeof(decltype(dev_image)::element_type),
                     width * sizeof(decltype(dev_image)::element_type),
                     height,
                     hipMemcpyHostToDevice);
        memcpy.record();
        hipDeviceSynchronize();
        cuda_throw_error();

        do_spotfinding_naive<<<blocks_dims, thread_block_size>>>(dev_image,
                                                                 dev_mask,
                                                                 width,
                                                                 height,
                                                                 result_sum,
                                                                 result_sumsq,
                                                                 result_n,
                                                                 result_strong);
        kernel.record();
        all.record();
        cuda_throw_error();
        hipDeviceSynchronize();

        print("    Read Time: \033[1m{:6.2f}\033[0m ms \033[37m({:.1f} GBps)\033[0m\n",
              start.elapsed_time(pre_load),
              GBps<pixel_t>(start.elapsed_time(pre_load), width * height));
        print("  Upload Time: \033[1m{:6.2f}\033[0m ms \033[37m({:.1f} GBps)\033[0m\n",
              memcpy.elapsed_time(start),
              GBps<pixel_t>(memcpy.elapsed_time(start), width * height));
        print("  Kernel Time: \033[1m{:6.2f}\033[0m ms\n", kernel.elapsed_time(memcpy));
        print("               ════════\n");
        print("        Total: \033[1m{:6.2f}\033[0m ms ({:.1f} GBps)\n",
              all.elapsed_time(pre_load),
              GBps<pixel_t>(all.elapsed_time(pre_load), width * height));

        auto strong =
          count_nonzero(result_strong.get(), width, height, device_mask_pitch);
        print("       Strong: {} px\n", strong);

        auto start_time = std::chrono::high_resolution_clock::now();
        size_t mismatch_x = 0, mismatch_y = 0;

        auto converted_image =
          std::vector<double>{host_image.get(), host_image.get() + width * height};
        auto dials_strong = spotfinder.standard_dispersion(
          converted_image, reader.get_mask().value_or(span<uint8_t>{}));
        auto end_time = std::chrono::high_resolution_clock::now();
        size_t dials_results = count_nonzero(dials_strong, width, height, width);

        print("        Dials: {} px\n", dials_results);
        bool validation_matches = compare_results(dials_strong.data(),
                                                  width,
                                                  result_strong.get(),
                                                  device_mask_pitch,
                                                  width,
                                                  height,
                                                  &mismatch_x,
                                                  &mismatch_y);
        float validation_time_ms =
          std::chrono::duration_cast<std::chrono::duration<double>>(end_time
                                                                    - start_time)
            .count()
          * 1000;

        if (validation_matches) {
            print("     Compared: \033[32mMatch\033[0m in {:.0f} ms\n",
                  validation_time_ms);
        } else {
            print("     Compared: \033[1;31mMismatch\033[0m in {:.0f} ms\n",
                  validation_time_ms);
            mismatch_x = max(static_cast<int>(mismatch_x) - 8, 0);
            mismatch_y = max(static_cast<int>(mismatch_y) - 8, 0);
            print("Data:\n");
            draw_image_data(host_image, mismatch_x, mismatch_y, 16, 16, width, height);
            print("Strong From DIALS:\n");
            draw_image_data(
              dials_strong, mismatch_x, mismatch_y, 16, 16, width, height);
            print("Strong From kernel:\n");
            draw_image_data(
              result_strong, mismatch_x, mismatch_y, 16, 16, device_mask_pitch, height);
            // print("Resultant N:\n");
            print("Sum From kernel:\n");
            draw_image_data(
              result_sum, mismatch_x, mismatch_y, 16, 16, device_pitch, height);
            print("Sum² From kernel:\n");
            draw_image_data(
              result_sumsq, mismatch_x, mismatch_y, 16, 16, device_pitch, height);
            print("Mask:\n");
            draw_image_data(reader.get_mask().value().data(),
                            mismatch_x,
                            mismatch_y,
                            16,
                            16,
                            width,
                            height);
        }
        print("Image:\n");
        draw_image_data(host_image, 0, 0, 16, 16, width, height);
        print("Sum:\n");
        draw_image_data(result_sum, 0, 0, 16, 16, device_pitch, height);
        print("SumSq:\n");
        draw_image_data(result_sumsq, 0, 0, 16, 16, device_pitch, height);
        print("N:\n");
        draw_image_data(result_n, 0, 0, 16, 16, device_mask_pitch, height);

        print("\n\n");
    }
}
